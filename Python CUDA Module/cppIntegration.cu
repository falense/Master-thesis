#include "hip/hip_runtime.h"


#define DEBUG false
#define BENCHMARK true
#define CPU_ENABLED false
#define GPU_ENABLED true

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <sys/time.h>
#include <iomanip>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"
//#include <helper_functions.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif




__device__ float devEuclidianDistanceSquared(const float x1,const  float y1,const  float x2,const  float y2){
		float t1 = x1-x2;
		float t2 = y1-y2;
		return t1*t1 + t2*t2; 
}


#define ALPHA 2.0f


__global__ void devCalculateQxy(float * __restrict__ resultGrid,  const float stepX, const float stepY,  const float *  __restrict__ samples, const unsigned int  sampleCount)
{
	const unsigned int resRowDim = blockDim.x * gridDim.x;
	const unsigned int resY = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int resX = blockIdx.x*blockDim.x + threadIdx.x;
	
    const unsigned int posX = resX*stepX;
    const unsigned int posY = resY*stepY;
    
	float ret = 0.0;
	for (unsigned int k = 0; k < (sampleCount); k++){
        float dk = devEuclidianDistanceSquared(posX,posY,samples[k*3+0],samples[k*3+1]);
        float Pk = samples[k*3+2];
        float lDk = log10(dk);
		for (unsigned int l = k+1; l < (sampleCount); l++){
            float dl = devEuclidianDistanceSquared(posX,posY,samples[l*3+0],samples[l*3+1]);
			float p = Pk-samples[l*3+2] - 5.0f * ALPHA * (log10(dl)-lDk);
			ret += p * p;
		}
	}
	
	resultGrid[resX+resRowDim*resY] = ret;
    return;
}

__global__ void devReduction(float *g_idata, float *g_odata, unsigned int * g_oindexdata, const unsigned int numElements){
	unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	unsigned int numThreads = blockDim.x*gridDim.x;
	
	unsigned int numElementsPerThread = numElements/numThreads;
	
	unsigned int blockIndex = blockDim.x * blockIdx.x * numElementsPerThread ;
	unsigned int startIndex = threadIdx.x * numElementsPerThread + blockIndex; 
	
	unsigned int bestIndex = startIndex;
	float bestScore = g_idata[startIndex];
	for (unsigned int i = startIndex; i < startIndex+numElementsPerThread; i++){
		if (g_idata[i] < bestScore){
			bestIndex = i;
			bestScore = g_idata[i];
		}
	}
		
	g_odata[tid] = bestScore;
	g_oindexdata[tid] = bestIndex;
	
	__syncthreads();
	
	unsigned int number = 1;
	
	while ( number <= (blockDim.x >> 1)){
		if (tid % (number*2) == 0){
			if (g_odata[tid] > g_odata[tid+number])
			{
				g_odata[tid] = g_odata[tid+number];
				g_oindexdata[tid] = g_oindexdata[tid+number];
			}
			
		}
		number <<= 1;
		
		__syncthreads();
	}
}



__global__ void devFinalReduction(float *g_odata, unsigned int * g_oindexdata, unsigned int stride){
	unsigned int numThreads = blockDim.x;
	unsigned int tid = threadIdx.x;
	
	unsigned int number = 1;
	
	while ( number <= (numThreads >> 1)){
		if (tid % (number*2) == 0){
			unsigned int firstIndex = stride*tid;
			unsigned int secondIndex = stride*(tid+number);
			if (g_odata[firstIndex] > g_odata[secondIndex])
			{
				g_odata[firstIndex] = g_odata[secondIndex];
				g_oindexdata[firstIndex] = g_oindexdata[secondIndex];
			}
		}
		number <<= 1;
		
		__syncthreads();
	}
}



inline float EuclidianDistanceSquared(const float x1,const  float y1,const  float x2,const  float y2){
		float t1 = x1-x2;
		float t2 = y1-y2;
		return t1*t1 + t2*t2; 
}


float Qxy(const float *pos, const float*  samples, const unsigned int sampleCount){
	float ret = 0.0;
	for (unsigned int k = 0; k < sampleCount; k++){
        float dk = EuclidianDistanceSquared(pos[0],pos[1],samples[k*3+0],samples[k*3+1]);
        
        
		for (unsigned int l = k+1; l < sampleCount; l++){
			//std::cout << "S:" << k << ":" << l << std::endl;
            float dl = EuclidianDistanceSquared(pos[0],pos[1],samples[l*3+0],samples[l*3+1]);
			float p = samples[k*3+2]-samples[l*3+2] - 5 * ALPHA * log10(dl/dk);
			ret += p * p;
		}
	}
	return ret;
}

void calculateQxy(float * resultGrid, const unsigned int * numSteps, const float * step, unsigned int  *bestPos, const float *samples, const unsigned int sampleCount)
{	
	
	float bestScore = 999999999.0;
	float pos[] = {0.0,0.0};
	for (int x = 0; x < numSteps[0]; x ++){
		//std::cout << "x:" << x ;
        pos[0] = x*step[0];
		for (int y = 0; y < numSteps[1]; y ++){
			//std::cout << " y:" << y << std::endl;
            pos[1] = y*step[1];
			float score = Qxy(pos,samples, sampleCount);
			if (score < bestScore){
				bestScore = score;
				bestPos[0] = x;
				bestPos[1] = y;
			}
			resultGrid[x+y*numSteps[0]] = score;
		}
	}
}

double gpuWrapperQxy(const dim3 & grid,const dim3  & threads,float * d_resultGrid, const float & stepX, const float & stepY,float * d_samples, const float * samples, const unsigned int & sampleCount){
	
    struct timeval t1, t2;
    double gpuTimeQxy;
    
    // Copy data to device
    checkCudaErrors(hipMemcpy(d_samples, samples, sizeof(float) * 3 * sampleCount,hipMemcpyHostToDevice));

    if (DEBUG)
		std::cout << std::endl << "Computing on GPU" << std::endl;
    
	if (BENCHMARK)
		gettimeofday(&t1, 0);
	
	devCalculateQxy<<< grid, threads >>>(d_resultGrid,stepX, stepY,d_samples,sampleCount);
    getLastCudaError("Kernel execution failed");
    
	checkCudaErrors(hipDeviceSynchronize());
	
	
	if (BENCHMARK){
		gettimeofday(&t2, 0);
		gpuTimeQxy = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec);
	}
	
    if (DEBUG){
		std::cout << "Done computing on GPU" << std::endl;
	
		if (BENCHMARK){
			printf("Time to generate:  %3.1f us \n", gpuTimeQxy);
		}
	}
	
	return gpuTimeQxy;
}

double cpuWrapperPredict(float * resultGridFromDevice, const float * d_resultGrid, const unsigned int gridMemSize, unsigned int * gpuBestPos, const unsigned int * numSteps, const float * step ){	
    struct timeval t1, t2;
    double cpuTimeReduction;
    
	if (DEBUG){
		std::cout << std::endl << "Calulating GPU prediction on CPU" << std::endl;
	}
    
    if (BENCHMARK)
		gettimeofday(&t1, 0);
	
    // copy results from device to host
    checkCudaErrors(hipMemcpy(resultGridFromDevice, d_resultGrid, gridMemSize, hipMemcpyDeviceToHost));
	
	float gpuBestScore = resultGridFromDevice[0];
	gpuBestPos[0] = 0;
	gpuBestPos[1] = 0;
	for (unsigned int x = 0; x < numSteps[0]; x++){
		for (unsigned int y = 0; y < numSteps[1]; y++){
			if (gpuBestScore > resultGridFromDevice[x+y*numSteps[0]]){
				gpuBestScore = resultGridFromDevice[x+y*numSteps[0]];
				gpuBestPos[0] = x;
				gpuBestPos[1] = y;
			}
				
		}
	}
	
	if (BENCHMARK){
		gettimeofday(&t2, 0);
		cpuTimeReduction = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec);	
	}
	
	if (DEBUG){
		std::cout <<  "GPU: Predicted position (" << gpuBestPos[0]*step[0] << "," << gpuBestPos[1]*step[1] << ")" <<  ", (x,y) = (" << gpuBestPos[0] << "," << gpuBestPos[1] << ")" << std::endl;
		std::cout << "Value: "  <<  gpuBestScore << std::endl;
		
		std::cout << "Finished calculating GPU prediction on CPU" << std::endl;
		

		if (BENCHMARK)
			printf("Time to generate:  %3.1f us \n", cpuTimeReduction);
	}
	return cpuTimeReduction;
}

double gpuWrapperPredict(float * d_resultGrid, const unsigned int gridSize, unsigned int *gpuBestPos, const unsigned int * numSteps, const float * step){	
	  
    struct timeval t1, t2;
    double gpuTimeReduction;
    
    float * d_maxResultGrid;
    float maxResult;
    unsigned int * d_maxIndexResultGrid;
    unsigned int maxIndexResult;
    
    
    checkCudaErrors(hipMalloc((void**)&d_maxResultGrid, sizeof(float)*gridSize));
    checkCudaErrors(hipMalloc((void**)&d_maxIndexResultGrid, sizeof(unsigned int)*gridSize));
	  
    dim3 gridPrediction(32,1, 1);
    dim3 threadsPrediction(1024, 1, 1);
    
	dim3 gridFinalReduction(1,1,1);
	dim3 blockFinalReduction(gridPrediction.x,1,1);
	
	if (DEBUG){
		std::cout << std::endl << "Calculating GPU prediction on GPU" << std::endl;
	}	
	
	if (BENCHMARK)
		gettimeofday(&t1, 0);
	
	devReduction<<< gridPrediction, threadsPrediction >>>(d_resultGrid,d_maxResultGrid,d_maxIndexResultGrid,gridSize);
    getLastCudaError("Kernel execution failed");
    
	checkCudaErrors(hipDeviceSynchronize());
	
	devFinalReduction<<< gridFinalReduction,blockFinalReduction>>>(d_maxResultGrid,d_maxIndexResultGrid,threadsPrediction.x);
    getLastCudaError("Kernel execution failed");
	
    checkCudaErrors(hipMemcpy(&maxIndexResult, d_maxIndexResultGrid, sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&maxResult, d_maxResultGrid, sizeof(float), hipMemcpyDeviceToHost));
    
    if (BENCHMARK){
		gettimeofday(&t2, 0);
		gpuTimeReduction = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec);
	}
	
	gpuBestPos[0] = maxIndexResult%numSteps[0];
	gpuBestPos[1] = maxIndexResult/numSteps[1];
	
	if (DEBUG){
		std::cout << "GPU reduction: (" << gpuBestPos[0]*step[0]  << "," << gpuBestPos[1]*step[1] << ")\t " <<  maxIndexResult<< std::endl;
		std::cout << "Value: "   << maxResult << std::endl;
		
		std::cout << "Finished calculating GPU prediction on GPU" << std::endl;
		if (BENCHMARK)
			printf("Time to generate:  %3.1f us \n", gpuTimeReduction);
	}
	
	checkCudaErrors(hipFree(d_maxResultGrid));
	checkCudaErrors(hipFree(d_maxIndexResultGrid));
	
	return gpuTimeReduction;
}

double cpuWrapperQxy(float * resultGrid, const unsigned int * numSteps, const float * step, unsigned int * cpuBestPos, const float * samples, const unsigned int sampleCount, const double cpuTimeReduction){
	struct timeval t1, t2;
    double cpuTimeQxy;
    
    if(DEBUG)
		std::cout << std::endl << "Computing on CPU" << std::endl;
    
	if (BENCHMARK)
		gettimeofday(&t1, 0);
	
	calculateQxy(resultGrid, numSteps, step, cpuBestPos, samples, sampleCount);
	
	if (BENCHMARK){
		gettimeofday(&t2, 0);
		cpuTimeQxy = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec) - cpuTimeReduction;
	}
	
    if(DEBUG){
		std::cout << "CPU: Predicted position (" << cpuBestPos[0]*step[0] << "," << cpuBestPos[1]*step[0] << ") , (x,y) = (" << cpuBestPos[0] << "," << cpuBestPos[1] << ")" << std::endl;
		std::cout << "Done computing on CPU" << std::endl;
		
		
		if(BENCHMARK){
			printf("Time to generate:  %3.1f us \n", cpuTimeReduction+cpuTimeQxy);
		}
 
	}
	
	return cpuTimeQxy;
}


void Qxy(float * resultGridFromDevice, float *samples, unsigned int * numSteps, float * gridMax, unsigned int numSamples){
	
	if (DEBUG){
		std::cout << "GridMax: (" << gridMax[0] << "," << gridMax[1] << ")" << std::endl;
		std::cout << "NumSteps: (" << numSteps[0] << "," << numSteps[1] << ")" << std::endl;
		std::cout << "NumSamples: " << numSamples << std::endl;
		
	}
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	findCudaDevice(0, 0);
	
	const float step [] = {gridMax[0]/numSteps[0],gridMax[1]/numSteps[1]};
		
	const unsigned int gridSize = numSteps[0] * numSteps[1];
	
	const unsigned int gridMemSize = gridSize * sizeof(float);
	const unsigned int sampleMemSize = sizeof(float) * 3 * numSamples;

	// Allocate device memory
	float *d_samples;
	float *d_resultGrid;
	
	checkCudaErrors(hipMalloc((void**)&d_samples, sampleMemSize));
	checkCudaErrors(hipMalloc((void**)&d_resultGrid, gridMemSize));
	
	// Setup grid and block sizes
	const unsigned int threadBlockDim = 16;
	dim3 grid(numSteps[0]/threadBlockDim, numSteps[1]/threadBlockDim, 1);
	dim3 threads(threadBlockDim, threadBlockDim, 1);
	
	std::cout << std::setprecision(3) << std::fixed;
	
	if (GPU_ENABLED)
		gpuWrapperQxy(grid,threads, d_resultGrid, step[0], step[1], d_samples, samples, numSamples);
	
	checkCudaErrors(hipMemcpy(resultGridFromDevice, d_resultGrid, gridMemSize, hipMemcpyDeviceToHost));
	
	
}


void predictSingle(float * result, int algorithm,  float *samples, unsigned int * numSteps,float * gridMax,unsigned int numSamples){
	if (algorithm == 0){
			if (DEBUG){
				std::cout << "Using algorithm: " << algorithm << std::endl;
				std::cout << "GridMax: (" << gridMax[0] << "," << gridMax[1] << ")" << std::endl;
				std::cout << "NumSteps: (" << numSteps[0] << "," << numSteps[1] << ")" << std::endl;
				std::cout << "NumSamples: " << numSamples << std::endl;
				
			}
			// use command-line specified CUDA device, otherwise use device with highest Gflops/s
			findCudaDevice(0, 0);
			
			const float step [] = {gridMax[0]/numSteps[0],gridMax[1]/numSteps[1]};
				
			const unsigned int gridSize = numSteps[0] * numSteps[1];
			
			const unsigned int gridMemSize = gridSize * sizeof(float);
			const unsigned int sampleMemSize = sizeof(float) * 3 * numSamples;

			unsigned int cpuCalcGpuBestPos [] = {0,0};
			unsigned int gpuBestPos [] = {0,0};
			unsigned int cpuBestPos[] = {0,0};
			
			double gpuTimeQxy;
			double gpuTimeReduction;
			double cpuTimeQxy;
			double cpuTimeReduction;
			
			float *resultGrid = (float*) malloc(gridMemSize);
			float *resultGridFromDevice = (float*) malloc(gridMemSize);
			
			// Allocate device memory
			float *d_samples;
			float *d_resultGrid;
			
			checkCudaErrors(hipMalloc((void**)&d_samples, sampleMemSize));
			checkCudaErrors(hipMalloc((void**)&d_resultGrid, gridMemSize));
			
			// Setup grid and block sizes
			const unsigned int threadBlockDim = 16;
			dim3 grid(numSteps[0]/threadBlockDim, numSteps[1]/threadBlockDim, 1);
			dim3 threads(threadBlockDim, threadBlockDim, 1);
			
			std::cout << std::setprecision(3) << std::fixed;
			
			if (GPU_ENABLED)
				gpuTimeQxy = gpuWrapperQxy(grid,threads, d_resultGrid, step[0], step[1], d_samples, samples, numSamples);
			
			if (CPU_ENABLED)
				cpuTimeReduction = cpuWrapperPredict(resultGridFromDevice, d_resultGrid, gridMemSize, cpuCalcGpuBestPos, numSteps, step);
			
			if (GPU_ENABLED)
				gpuTimeReduction = gpuWrapperPredict(d_resultGrid, gridSize, gpuBestPos, numSteps, step);
			
			if (CPU_ENABLED)
				cpuTimeQxy = cpuWrapperQxy(resultGrid, numSteps, step,  cpuBestPos, samples, numSamples, cpuTimeReduction);
			
			if (DEBUG && CPU_ENABLED && GPU_ENABLED){
				float error = sqrt(pow(cpuBestPos[0]-gpuBestPos[0],2)+pow(cpuBestPos[1]-gpuBestPos[1],2));
				std::cout << "GPU vs CPU error was: " << error << std::endl;
		
				if(BENCHMARK){
					printf("\n\t\t Qxy \t\t Reduction \t\t Qxy + Reduction \n");
					printf("CPU: \t %10.1f \t\t %10.1f \t\t %10.1f \n", cpuTimeQxy, cpuTimeReduction, cpuTimeQxy+cpuTimeReduction);
					printf("GPU: \t %10.1f \t\t %10.1f \t\t %10.1f \n", gpuTimeQxy, gpuTimeReduction, gpuTimeQxy+gpuTimeReduction);
					printf("Speedup: %10.1f x \t\t %10.1f x \t\t %10.1f x \n", cpuTimeQxy/gpuTimeQxy, cpuTimeReduction/gpuTimeReduction, (cpuTimeQxy+cpuTimeReduction) / (gpuTimeQxy+gpuTimeReduction));
				}
			}

		    checkCudaErrors(hipFree(d_samples));
		    checkCudaErrors(hipFree(d_resultGrid));
			free(resultGrid);
			free(resultGridFromDevice);
			
			result[0] = gpuBestPos[0]*step[0];
			result[1] = gpuBestPos[1]*step[1];
	}
	
}

void predict(float * result, int algorithm,  float *samples,unsigned int * numSteps,float * gridMax,unsigned int numSamples,unsigned int numSets){
	
	for ( unsigned int i = 0; i < numSets; i++){
		/*for (unsigned int j = 0; j < numSamples; j++){
			for (unsigned int l = 0; l < 3; l++){
				std::cout << samples[i*numSamples*3 + j*3 + l] << std::endl;
			}
			std::cout << std::endl;
		}*/
		predictSingle(&result[i*2], algorithm, &samples[i*numSamples*3], numSteps, gridMax, numSamples);
	}
}

extern "C" bool
runTest(int argc, char **argv)
{
	/*	float * samples = (float*) malloc(sampleMemSize);
			srand(11);
			for (unsigned int i = 0; i < sampleCount; i++){
				samples[i*3+0] = (float) (rand()%100);
				samples[i*3+1] = (float) (rand()%100);
				samples[i*3+2] = (float) (rand()%10);
				
			}*/
				float samples2 [] = {
			103.044449975,37.5749896125,-26.1923435464 ,
			103.026091643,37.523839946,-25.5523156053 ,
			103.994168647,36.9311709869,-26.4742067687 ,
			106.180122539,35.9270807992,-25.8361988542 ,
			109.290196049,34.5971894763,-25.8526809338 ,
			115.946530409,32.049713707,-26.6842774032 ,
			127.328613766,28.4147515553,-29.6847952096 ,
			133.875968633,24.6830498173,-35.7722972657 ,
			143.106648943,21.1359316569,-30.2415020612 ,
			150.743984466,16.3434303186,-26.6873405738 ,
			156.203120413,13.1654793153,-24.9629362518 ,
			161.974391634,9.81628812497,-28.1841922118 ,
			164.702297142,8.26956669457,-31.4362932872 ,
			166.627383183,6.56050067206,-38.8563342597 ,
			167.110314051,6.35367810877,-34.6971965389 ,
			167.729573226,5.71319533124,-34.1375850024 ,
			167.722353599,5.42631242046,-33.5265873133 ,
			167.433578103,5.95337637291,-35.3004719541 ,
			169.510981532,7.02751936468,-33.9005561564 ,
			173.567809903,10.3500237728,-38.4458296544 ,
			177.61271973,18.391640867,-29.7300924561 ,
			185.796451677,24.6196687086,-27.1066268752 ,
			192.341180063,29.8280390729,-35.5981976925 ,
			196.504742503,33.9144526271,-41.8390766975 ,
			200.716080698,39.1317185847,-33.8883913917 ,
			204.907923252,44.5513593098,-34.393778751 ,
			208.625755546,49.6852290727,-31.4775922602 ,
			214.598995506,55.5140671278,-32.6440687182 ,
			219.646438802,61.4529881598,-34.8694172927 ,
			221.884176772,69.2444166698,-50.3789218541 ,
			219.220136902,73.0739699434,-32.4132878969 ,
			214.978122963,79.8068227518,-32.973072406 ,
			211.640281516,84.9784987897,-34.0524302717 ,
			207.944929,86.4029058002,-29.0272555731 ,
			201.613542127,88.8547539325,-31.2348066541 ,
			191.571280357,92.0727351096,-31.3896116721 ,
			180.84536686,93.6194565392,-29.8945293508 ,
			169.318330547,94.9715868477,-29.4668411486 ,
			157.960500687,94.731405806,-27.6718815559 ,
			145.938924215,95.0494232965,-34.2399422316 ,
			135.981922617,96.2781272356,-31.8002077752 ,
			129.785256945,98.155097597,-29.1874275027 ,
			124.965623704,98.0516863154,-24.0942554996 ,
			121.237590584,96.5238680233,-24.0951838818 ,
			117.934004739,96.3381724962,-25.8207170094 ,
			115.948451371,94.7136146179,-24.0655931066 ,
			114.231016279,94.3700222945,-25.4052220419 ,
			113.483327878,92.9700781676,-23.1736302667 ,
			113.246323658,92.9678542697,-23.6323510354 ,
			112.904172342,92.8777863788,-24.0816911862 ,
			112.943140213,92.111653334,-23.5958904402 ,
			113.162364073,91.3744309702,-24.5591877125 ,
			111.596895885,88.2064875103,-24.5716028518 ,
			110.154366518,85.4288382428,-25.623572934 ,
			108.216142632,82.1341325663,-24.4579167494 ,
			106.164978873,78.7460231516,-22.6656360697 ,
			102.629520311,73.4642641358,-22.1670262145 ,
			96.7474356914,65.2847653322,-21.4916445096 ,
			91.9635027205,59.6694215366,-20.6601490868 ,
			86.9158408255,54.1619368193,-20.5624620322 ,
			83.9800378138,50.7938424915,-21.5829190437 ,
			83.1094638584,45.4431426218,-21.2738889449 ,
			83.8423017339,40.2625709891,-19.5858379141 ,
			84.3486527729,37.3370324687,-20.2683026221 ,
			88.8323866668,34.9808119733,-20.2249963105 ,
			92.7386020518,33.635353361,-20.9845766019 ,
			97.8120674029,31.9296231866,-20.1387172421 ,
			100.885385259,31.9285112372,-21.2429675891 ,
			102.674659421,30.8877267238,-22.3030164832 ,
			102.357506445,31.9385187807,-24.1537852405 ,
			102.62176086,32.4878217179,-22.7039022625 ,
			103.274914458,32.836973788,-22.7135212491 ,
			104.02206951,33.9922890759,-21.9097440792 ,
			104.082112026,35.5501299983,-27.3000627279 ,
			103.863993038,36.5119661135,-22.2377354043 ,
			103.910146378,37.3715028965,-24.3208055833 ,
			103.901780884,38.0853743254,-26.0922559655 ,
			103.956840543,38.7803426167,-20.5548585865 ,
			103.911209788,39.118375194,-25.031573264 ,
			103.817730276,39.5320203214,-24.5408298697 ,
			104.352389753,39.4897662489,-22.2280253548 ,
			104.542110903,39.3596681851,-23.1608972453 ,
			104.70957962,39.1661890125,-22.7137658272 ,
			104.720704468,39.24624936,-22.9558318635
			};

			unsigned int numSamples = 40;
			float * samples = &samples2[0];
			
			const unsigned int magnitude = 10;
			
			unsigned int numSteps[] = {(1 << magnitude),(1 << magnitude)};
			float gridMax[] = {100.0,100.0};
			
			int algorithm  = 0;
			
			float * result = (float*) malloc(sizeof(float)*2);
			predictSingle(result,algorithm,  samples, numSteps, gridMax, numSamples);

    return true;
}
